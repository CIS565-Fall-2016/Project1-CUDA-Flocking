#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <glm/glm.hpp>
#include "utilityCore.hpp"
#include "kernel.h"
#include ""

// LOOK-2.1 potentially useful for doing grid-based neighbor search
#ifndef imax
#define imax( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef imin
#define imin( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#ifndef clamp
#define clamp(x, lo, hi) (x < lo) ? lo : (x > hi) ? hi : x
#endif

#ifndef wrap
#define wrap(x, lo, hi) (x < lo) ? x + (hi - lo) : (x > hi) ? x - (hi - lo) : x
#endif

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

/**
* Check for CUDA errors; print and exit if there was a problem.
*/
void checkCUDAError(const char *msg, int line = -1) {
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    if (line >= 0) {
      fprintf(stderr, "Line %d: ", line);
    }
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}


/*****************
* Configuration *
*****************/

/*! Block size used for CUDA kernel launch. */
#define blockSize 128

// LOOK-1.2 Parameters for the boids algorithm.
// These worked well in our reference implementation.
#define rule1Distance 5.0f
#define rule2Distance 3.0f
#define rule3Distance 5.0f

#define rule1Scale 0.01f
#define rule2Scale 0.1f
#define rule3Scale 0.1f

#define maxSpeed 1.0f

/*! Size of the starting area in simulation space. */
#define scene_scale 100.0f

//#define gridFactor 2.0f

/***********************************************
* Kernel state (pointers are device pointers) *
***********************************************/

int numObjects;
dim3 threadsPerBlock(blockSize);

// LOOK-1.2 - These buffers are here to hold all your boid information.
// These get allocated for you in Boids::initSimulation.
// Consider why you would need two velocity buffers in a simulation where each
// boid cares about its neighbors' velocities.
// These are called ping-pong buffers.
glm::vec3 *dev_pos;
glm::vec3 *dev_vel1;
glm::vec3 *dev_vel2;

// LOOK-2.1 - these are NOT allocated for you. You'll have to set up the thrust
// pointers on your own too.

// For efficient sorting and the uniform grid. These should always be parallel.
int *dev_particleArrayIndices; // What index in dev_pos and dev_velX represents this particle?
int *dev_particleGridIndices; // What grid cell is this particle in?
// needed for use with thrust
thrust::device_ptr<int> dev_thrust_particleArrayIndices;
thrust::device_ptr<int> dev_thrust_particleGridIndices;

int *dev_gridCellStartIndices; // What part of dev_particleArrayIndices belongs
int *dev_gridCellEndIndices;   // to this cell?

// TODO-2.3 - consider what additional buffers you might need to reshuffle
// the position and velocity data to be coherent within cells.
glm::vec3 *dev_pos_sorted;
glm::vec3 *dev_vel_sorted;

// LOOK-2.1 - Grid parameters based on simulation parameters.
// These are automatically computed for you in Boids::initSimulation
int gridCellCount;
int gridSideCount;
float gridCellWidth;
float gridInverseCellWidth;
glm::vec3 gridMinimum;

/******************
* initSimulation *
******************/

__host__ __device__ unsigned int hash(unsigned int a) {
  a = (a + 0x7ed55d16) + (a << 12);
  a = (a ^ 0xc761c23c) ^ (a >> 19);
  a = (a + 0x165667b1) + (a << 5);
  a = (a + 0xd3a2646c) ^ (a << 9);
  a = (a + 0xfd7046c5) + (a << 3);
  a = (a ^ 0xb55a4f09) ^ (a >> 16);
  return a;
}

/**
* LOOK-1.2 - this is a typical helper function for a CUDA kernel.
* Function for generating a random vec3.
*/
__host__ __device__ glm::vec3 generateRandomVec3(float time, int index) {
  thrust::default_random_engine rng(hash((int)(index * time)));
  thrust::uniform_real_distribution<float> unitDistrib(-1, 1);

  return glm::vec3((float)unitDistrib(rng), (float)unitDistrib(rng), (float)unitDistrib(rng));
}

/**
* LOOK-1.2 - This is a basic CUDA kernel.
* CUDA kernel for generating boids with a specified mass randomly around the star.
*/
__global__ void kernGenerateRandomPosArray(int time, int N, glm::vec3 * arr, float scale) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    glm::vec3 rand = generateRandomVec3(time, index);
    arr[index].x = scale * rand.x;
    arr[index].y = scale * rand.y;
    arr[index].z = scale * rand.z;
  }
}

/**
* Initialize memory, update some globals
*/
void Boids::initSimulation(int N) {
  numObjects = N;
  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  // LOOK-1.2 - This is basic CUDA memory management and error checking.
  // Don't forget to hipFree in  Boids::endSimulation.
  hipMalloc((void**)&dev_pos, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_pos failed!");

  hipMalloc((void**)&dev_vel1, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel1 failed!");
	
  hipMalloc((void**)&dev_vel2, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel2 failed!");
	
  // LOOK-1.2 - This is a typical CUDA kernel invocation.
  kernGenerateRandomPosArray<<<fullBlocksPerGrid, blockSize>>>(1, numObjects,
    dev_pos, scene_scale);
  checkCUDAErrorWithLine("kernGenerateRandomPosArray failed!");

  // LOOK-2.1 computing grid params
	gridCellWidth = 2.0f * std::max(std::max(rule1Distance, rule2Distance), rule3Distance);
  int halfSideCount = (int)(scene_scale / gridCellWidth) + 1;
  gridSideCount = 2 * halfSideCount;

  gridCellCount = gridSideCount * gridSideCount * gridSideCount;
  gridInverseCellWidth = 1.0f / gridCellWidth;
  float halfGridWidth = gridCellWidth * halfSideCount;
  gridMinimum.x -= halfGridWidth;
  gridMinimum.y -= halfGridWidth;
  gridMinimum.z -= halfGridWidth;

  // TODO-2.1 TODO-2.3 - Allocate additional buffers here.
  hipMalloc((void**)&dev_particleArrayIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleArrayIndices failed!");
	
  hipMalloc((void**)&dev_particleGridIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleGridIndices failed!");

  hipMalloc((void**)&dev_gridCellStartIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellStartIndices failed!");
	
  hipMalloc((void**)&dev_gridCellEndIndices, gridCellCount * sizeof(int));
	checkCUDAErrorWithLine("hipMalloc dev_gridCellEndIndices failed!");

	hipMalloc((void**)&dev_pos_sorted, N * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMalloc dev_pos_sorted failed!");

	hipMalloc((void**)&dev_vel_sorted, N * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMalloc dev_vel_sorted failed!");

  hipDeviceSynchronize();
}


/******************
* copyBoidsToVBO *
******************/

/**
* Copy the boid positions into the VBO so that they can be drawn by OpenGL.
*/
__global__ void kernCopyPositionsToVBO(int N, glm::vec3 *pos, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  float c_scale = -1.0f / s_scale;

  if (index < N) {
    vbo[4 * index + 0] = pos[index].x * c_scale;
    vbo[4 * index + 1] = pos[index].y * c_scale;
    vbo[4 * index + 2] = pos[index].z * c_scale;
    vbo[4 * index + 3] = 1.0f;
  }
}

__global__ void kernCopyVelocitiesToVBO(int N, glm::vec3 *vel, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  if (index < N) {
    vbo[4 * index + 0] = vel[index].x + 0.3f;
    vbo[4 * index + 1] = vel[index].y + 0.3f;
    vbo[4 * index + 2] = vel[index].z + 0.3f;
    vbo[4 * index + 3] = 1.0f;
  }
}

/**
* Wrapper for call to the kernCopyboidsToVBO CUDA kernel.
*/
void Boids::copyBoidsToVBO(float *vbodptr_positions, float *vbodptr_velocities) {
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

  kernCopyPositionsToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_pos, vbodptr_positions, scene_scale);
  kernCopyVelocitiesToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_vel1, vbodptr_velocities, scene_scale);

  checkCUDAErrorWithLine("copyBoidsToVBO failed!");

  hipDeviceSynchronize();
}


/******************
* stepSimulation *
******************/


/**
* Helper function to compute unscaled velocity update for a single pair of boids
*/
__device__ int computeVelocityChangePair(
	const glm::vec3 &pos1, const glm::vec3 &vel1, 
	const glm::vec3 &pos2, const glm::vec3 &vel2,
	glm::vec3 &cen, glm::vec3 &sep, glm::vec3 &coh ) {

	int n = 0;
	glm::vec3 dv;

	// RULE 1:  Move to center of mass
	float dist = glm::length(pos2 - pos1);
	if (dist < rule1Distance) {
		cen += pos2;
		n++;
	}

	// RULE 2:  Maintain minimum distance from neighbors
	if (dist < rule2Distance) {
		sep -= (pos2 - pos1);
	}

	// RULE 3:  Align Velocities
	if (dist < rule3Distance) {
		coh += vel2;
	}
	
  return n;
}

/**
* LOOK-1.2 You can use this as a helper for kernUpdateVelocityBruteForce.
* __device__ code can be called from a __global__ context
* Compute the new velocity on the body with index `iSelf` due to the `N` boids
* in the `pos` and `vel` arrays.
*/
__device__ glm::vec3 computeVelocityChange(int N, int iSelf, const glm::vec3 *pos, const glm::vec3 *vel) {

	glm::vec3 dv = glm::vec3(0.0);
	glm::vec3 center = glm::vec3(0.0);
	glm::vec3 separate = glm::vec3(0.0);
	glm::vec3 cohesion = glm::vec3(0.0);
	float nBoids = 0;

  for(int i = 0; i < N; i++) {
		if (i != iSelf) {
			nBoids += computeVelocityChangePair(pos[iSelf], vel[iSelf], pos[i], vel[i], center, separate, cohesion) ;
		}
  }

	if (nBoids > 0) {
		center /= nBoids;
		dv = (center - pos[iSelf]) * rule1Scale + cohesion * rule3Scale + separate * rule2Scale;
	}

  return dv;
}

/**
* TODO-1.2 implement basic flocking
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdateVelocityBruteForce(int N, glm::vec3 *pos,
  glm::vec3 *vel1, glm::vec3 *vel2) {
  // Compute a new velocity based on pos and vel1
  // Clamp the speed
  // Record the new velocity into vel2. Question: why NOT vel1?

  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }

	glm::vec3 dv = computeVelocityChange(N, index, pos, vel1);
	glm::vec3 newVel = glm::vec3(0.0);
	
	newVel = vel2[index] + dv;
	float speed = glm::length(newVel);
	vel2[index] = (speed <= maxSpeed) ? newVel : newVel * maxSpeed / speed;
}

/**
* LOOK-1.2 Since this is pretty trivial, we implemented it for you.
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdatePos(int N, float dt, glm::vec3 *pos, glm::vec3 *vel) {
  // Update position by velocity
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }
  glm::vec3 thisPos = pos[index];
  thisPos += vel[index] * dt;

  // Wrap the boids around so we don't lose them
	float wrapScale = 1.0f;
  thisPos.x = thisPos.x < -scene_scale ? scene_scale * wrapScale : thisPos.x;
  thisPos.y = thisPos.y < -scene_scale ? scene_scale * wrapScale : thisPos.y;
  thisPos.z = thisPos.z < -scene_scale ? scene_scale * wrapScale : thisPos.z;

  thisPos.x = thisPos.x > scene_scale ? -scene_scale * wrapScale : thisPos.x;
  thisPos.y = thisPos.y > scene_scale ? -scene_scale * wrapScale : thisPos.y;
  thisPos.z = thisPos.z > scene_scale ? -scene_scale * wrapScale : thisPos.z;

  pos[index] = thisPos;
}

// LOOK-2.1 Consider this method of computing a 1D index from a 3D grid index.
// LOOK-2.3 Looking at this method, what would be the most memory efficient
//          order for iterating over neighboring grid cells?
//          for(x)
//            for(y)
//             for(z)? Or some other order?
__device__ int gridIndex3Dto1D(int x, int y, int z, int gridResolution) {
  return x + y * gridResolution + z * gridResolution * gridResolution;
}

__global__ void kernComputeIndices(int N, int gridResolution,
  glm::vec3 gridMin, float inverseCellWidth,
  glm::vec3 *pos, int *indices, int *gridIndices) {
  // TODO-2.1
  // - Label each boid with the index of its grid cell.
  // - Set up a parallel array of integer indices as pointers to the actual
  //   boid data in pos and vel1/vel2

	int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }
	
	// calculate the index from grid dimensions
	glm::vec3 temp = (pos[index] - gridMin) * inverseCellWidth;
	gridIndices[index] = gridIndex3Dto1D((int) temp.x, (int) temp.y, (int) temp.z, gridResolution);

	// for sorting a dictionary
	indices[index] = index;
}

// LOOK-2.1 Consider how this could be useful for indicating that a cell
//          does not enclose any boids
__global__ void kernResetIntBuffer(int N, int *intBuffer, int value) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    intBuffer[index] = value;
  }
}

__global__ void kernIdentifyCellStartEnd(int N, int *particleGridIndices,
  int *gridCellStartIndices, int *gridCellEndIndices) {
  // TODO-2.1
  // Identify the start point of each cell in the gridIndices array.
  // This is basically a parallel unrolling of a loop that goes
  // "this index doesn't match the one before it, must be a new cell!"
			
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= N) {
    return;
  }

	int Cell = particleGridIndices[index];

	if (index <= 0)	{
		gridCellStartIndices[Cell] = index;
			
		if (particleGridIndices[index + 1] != Cell)
			gridCellEndIndices[Cell] = index;
	} else if (index >= (N-1)) {
		if (particleGridIndices[index - 1] != Cell)
			gridCellStartIndices[Cell] = index;

		gridCellEndIndices[Cell] = index;
	} else {
		if (particleGridIndices[index - 1] != Cell)
			gridCellStartIndices[Cell] = index;
			
		if (particleGridIndices[index + 1] != Cell)
			gridCellEndIndices[Cell] = index;
	}

}

/**
* Helper function to find neighboring grid cells to search
*/
__device__ void computeNeighborList(int gridResolution, float cellWidth, glm::vec3 &pos, int *Neighbors ) {

	int gridCount = gridResolution*gridResolution*gridResolution;
	
	// x-axis
	if (std::fmod(pos.x, cellWidth) < cellWidth/2) {	
		Neighbors[1] = wrap(Neighbors[0] - 1, 0, gridCount);
	} else {
		Neighbors[1] = wrap(Neighbors[0] + 1, 0, gridCount);	
	}

	// y-axis
	if (std::fmod(pos.y, cellWidth) < cellWidth/2) {
		Neighbors[2] = wrap(Neighbors[0] - gridResolution, 0, gridCount);
		Neighbors[3] = wrap(Neighbors[1] - gridResolution, 0, gridCount);
	} else {
		Neighbors[2] = wrap(Neighbors[0] + gridResolution, 0, gridCount);
		Neighbors[3] = wrap(Neighbors[1] + gridResolution, 0, gridCount);
	}

	// z-axis	
	if (std::fmod(pos.z, cellWidth) < cellWidth/2) {
		Neighbors[4] = wrap(Neighbors[0] - gridResolution*gridResolution, 0, gridCount);
		Neighbors[5] = wrap(Neighbors[1] - gridResolution*gridResolution, 0, gridCount);
		Neighbors[6] = wrap(Neighbors[2] - gridResolution*gridResolution, 0, gridCount);
		Neighbors[7] = wrap(Neighbors[3] - gridResolution*gridResolution, 0, gridCount);
	} else {
		Neighbors[4] = wrap(Neighbors[0] + gridResolution*gridResolution, 0, gridCount);
		Neighbors[5] = wrap(Neighbors[1] + gridResolution*gridResolution, 0, gridCount);
		Neighbors[6] = wrap(Neighbors[2] + gridResolution*gridResolution, 0, gridCount);
		Neighbors[7] = wrap(Neighbors[3] + gridResolution*gridResolution, 0, gridCount);
	}

}


__global__ void kernUpdateVelNeighborSearchScattered(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  int *particleArrayIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  // TODO-2.1 - Update a boid's velocity using the uniform grid to reduce
  // the number of boids that need to be checked.
  // - Identify the grid cell that this particle is in
  // - Identify which cells may contain neighbors. This isn't always 8.
  // - For each cell, read the start/end indices in the boid pointer array.
  // - Access each boid in the cell and compute velocity change from
  //   the boids rules, if this boid is within the neighborhood distance.
  // - Clamp the speed change before putting the new speed in vel2
				
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= N) {
    return;
  }
	
	// Current Cell Index
	glm::vec3 temp = (pos[index] - gridMin) * inverseCellWidth;
	int CurrentCell = gridIndex3Dto1D((int) temp.x, (int) temp.y, (int) temp.z, gridResolution);

	// Neighboring Cells
	int Neighbors[8] = {CurrentCell};
	computeNeighborList(gridResolution, cellWidth, pos[index], Neighbors);

	// check all boids in neighboring cells
	glm::vec3 dv = glm::vec3(0.0);
	glm::vec3 center = glm::vec3(0.0);
	glm::vec3 separate = glm::vec3(0.0);
	glm::vec3 cohesion = glm::vec3(0.0);
	float nBoids = 0.0;

	for (int i = 0; i < 8; i++) {
		if (gridCellStartIndices[Neighbors[i]] != -1) {
			for (int j = gridCellStartIndices[Neighbors[i]]; j <= gridCellEndIndices[Neighbors[i]]; j++) {
				nBoids += computeVelocityChangePair(pos[index], vel1[index], pos[particleArrayIndices[j]], vel1[particleArrayIndices[j]], center, separate, cohesion);
			}
		}
	}

	if (nBoids > 0) {
		center /= nBoids;
		dv = (center - pos[index]) * rule1Scale + cohesion * rule3Scale + separate * rule2Scale;
	}
	
	// calculate new velocity and clamp
	glm::vec3 newVel = vel2[index] + dv;
	float speed = glm::length(newVel);
	vel2[index] = (speed <= maxSpeed) ? newVel : newVel * maxSpeed / speed;
}

__global__ void kernUpdateVelNeighborSearchCoherent(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
	glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2,
	glm::vec3 *pos_sort, glm::vec3 *vel_sort){
  // TODO-2.3 - This should be very similar to kernUpdateVelNeighborSearchScattered,
  // except with one less level of indirection.
  // This should expect gridCellStartIndices and gridCellEndIndices to refer
  // directly to pos and vel1.
  // - Identify the grid cell that this particle is in
  // - Identify which cells may contain neighbors. This isn't always 8.
  // - For each cell, read the start/end indices in the boid pointer array.
  //   DIFFERENCE: For best results, consider what order the cells should be
  //   checked in to maximize the memory benefits of reordering the boids data.
  // - Access each boid in the cell and compute velocity change from
  //   the boids rules, if this boid is within the neighborhood distance.
  // - Clamp the speed change before putting the new speed in vel2
					
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= N) {
    return;
  }
	
	// Current Cell Index
	glm::vec3 temp = (pos[index] - gridMin) * inverseCellWidth;
	int CurrentCell = gridIndex3Dto1D((int) temp.x, (int) temp.y, (int) temp.z, gridResolution);

	// Neighboring Cells
	int Neighbors[8] = {CurrentCell};
	computeNeighborList(gridResolution, cellWidth, pos[index], Neighbors);

	// check all boids in neighboring cells
	glm::vec3 dv = glm::vec3(0.0);
	glm::vec3 center = glm::vec3(0.0);
	glm::vec3 separate = glm::vec3(0.0);
	glm::vec3 cohesion = glm::vec3(0.0);
	float nBoids = 0.0;

	for (int i = 0; i < 8; i++) {
		if (gridCellStartIndices[Neighbors[i]] != -1) {
			for (int j = gridCellStartIndices[Neighbors[i]]; j <= gridCellEndIndices[Neighbors[i]]; j++) {
				nBoids += computeVelocityChangePair(pos[index], vel1[index], pos_sort[j], vel_sort[j], center, separate, cohesion);
			}
		}
	}

	if (nBoids > 0) {
		center /= nBoids;
		dv = (center - pos[index]) * rule1Scale + cohesion * rule3Scale + separate * rule2Scale;
	}
	
	// calculate new velocity and clamp
	glm::vec3 newVel = vel2[index] + dv;
	float speed = glm::length(newVel);
	vel2[index] = (speed <= maxSpeed) ? newVel : newVel * maxSpeed / speed;
}

// Use this to pull velocity and position data out of the Wrapper container
__global__ void kernSortArray(int N, int *indices, glm::vec3 *vec1, glm::vec3 *vec2) {
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= N) {
		return;
	}

	vec2[index] = vec1[indices[index]];
}

/**
* Step the entire N-body simulation by `dt` seconds.
*/
void Boids::stepSimulationNaive(float dt) {
  // TODO-1.2 - use the kernels you wrote to step the simulation forward in time.
  // TODO-1.2 ping-pong the velocity buffers
	
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
	
	// Swap Velocity Buffers
	glm::vec3 *temp = dev_vel1;
	dev_vel1 = dev_vel2;
	dev_vel2 = temp;

	// Update Velocities and Positions
	kernUpdateVelocityBruteForce<<<fullBlocksPerGrid, blockSize>>>(numObjects, dev_pos, dev_vel1, dev_vel2);
	kernUpdatePos<<<fullBlocksPerGrid, blockSize>>>(numObjects, dt, dev_pos, dev_vel2);
}

void Boids::stepSimulationScatteredGrid(float dt) {
  // TODO-2.1
  // Uniform Grid Neighbor search using Thrust sort.
  // In Parallel:
  // - label each particle with its array index as well as its grid index.
  //   Use 2x width grids.
  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
  // - Perform velocity updates using neighbor search
  // - Update positions
  // - Ping-pong buffers as needed

	dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

	// Swap Velocity Buffers
	hipMemcpy(dev_vel1, dev_vel2, numObjects * sizeof(glm::vec3), hipMemcpyDeviceToDevice);

	// Bin the Boids
	kernComputeIndices<<<fullBlocksPerGrid, blockSize>>>(numObjects, gridSideCount,  gridMinimum, gridInverseCellWidth, dev_pos, dev_particleArrayIndices, dev_particleGridIndices);
	
	// Sort
	thrust::device_ptr<int> dev_thrust_keys(dev_particleGridIndices);		
  thrust::device_ptr<int> dev_thrust_values(dev_particleArrayIndices);
  thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + numObjects, dev_thrust_values);

	// Calculate boids in each cell
	kernResetIntBuffer << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_gridCellStartIndices, -1);
	kernResetIntBuffer << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_gridCellEndIndices, -1);
	kernIdentifyCellStartEnd << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);

	// Update Velocities and Positions
	kernUpdateVelNeighborSearchScattered<<<fullBlocksPerGrid, blockSize>>>(
		numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, gridCellWidth,
		dev_gridCellStartIndices, dev_gridCellEndIndices,	dev_particleArrayIndices,
		dev_pos, dev_vel1, dev_vel2);

	kernUpdatePos<<<fullBlocksPerGrid, blockSize>>>(numObjects, dt, dev_pos, dev_vel2);
}

void Boids::stepSimulationCoherentGrid(float dt) {
  // TODO-2.3 - start by copying Boids::stepSimulationNaiveGrid
  // Uniform Grid Neighbor search using Thrust sort on cell-coherent data.
  // In Parallel:
  // - Label each particle with its array index as well as its grid index.
  //   Use 2x width grids
  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
  // - BIG DIFFERENCE: use the rearranged array index buffer to reshuffle all
  //   the particle data in the simulation array.
  //   CONSIDER WHAT ADDITIONAL BUFFERS YOU NEED
  // - Perform velocity updates using neighbor search
  // - Update positions
  // - Ping-pong buffers as needed. THIS MAY BE DIFFERENT FROM BEFORE.


	dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

	// Swap Velocity Buffers
	hipMemcpy(dev_vel1, dev_vel2, numObjects * sizeof(glm::vec3), hipMemcpyDeviceToDevice);

	// Bin the Boids
	kernComputeIndices<<<fullBlocksPerGrid, blockSize>>>(numObjects, gridSideCount,  gridMinimum, gridInverseCellWidth, dev_pos, dev_particleArrayIndices, dev_particleGridIndices);
		
	// Sort	
	thrust::device_ptr<int> dev_thrust_keys(dev_particleGridIndices);
	thrust::device_ptr<int> dev_thrust_values(dev_particleArrayIndices);
	thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + numObjects, dev_thrust_values);
	
	// Reoder arrays
	kernSortArray << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_particleArrayIndices, dev_vel1, dev_vel_sorted);
	kernSortArray << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_particleArrayIndices, dev_pos, dev_pos_sorted);

	// Calculate boids in each cell
	kernResetIntBuffer << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_gridCellStartIndices, -1);
	kernResetIntBuffer << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_gridCellEndIndices, -1);
	kernIdentifyCellStartEnd << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);

	// Update Velocities and Positions
	kernUpdateVelNeighborSearchCoherent<<<fullBlocksPerGrid, blockSize>>>(
		numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, gridCellWidth,
		dev_gridCellStartIndices, dev_gridCellEndIndices,	dev_pos, dev_vel1, dev_vel2, dev_pos_sorted, dev_vel_sorted);

	kernUpdatePos<<<fullBlocksPerGrid, blockSize>>>(numObjects, dt, dev_pos, dev_vel2);
}

void Boids::endSimulation() {
  hipFree(dev_vel1);
  hipFree(dev_vel2);
  hipFree(dev_pos);

  // TODO-2.1 TODO-2.3 - Free any additional buffers here.
	hipFree(dev_particleArrayIndices);
  hipFree(dev_particleGridIndices);
	hipFree(dev_gridCellStartIndices);
	hipFree(dev_gridCellEndIndices);
	hipFree(dev_pos_sorted);
	hipFree(dev_vel_sorted);
}

void Boids::unitTest() {
  // LOOK-1.2 Feel free to write additional tests here.

  // test unstable sort
  int *dev_intKeys;
  int *dev_intValues;
  int N = 10;

  int *intKeys = new int[N];
  int *intValues = new int[N];

  intKeys[0] = 0; intValues[0] = 0;
  intKeys[1] = 1; intValues[1] = 1;
  intKeys[2] = 0; intValues[2] = 2;
  intKeys[3] = 3; intValues[3] = 3;
  intKeys[4] = 0; intValues[4] = 4;
  intKeys[5] = 2; intValues[5] = 5;
  intKeys[6] = 2; intValues[6] = 6;
  intKeys[7] = 0; intValues[7] = 7;
  intKeys[8] = 5; intValues[8] = 8;
  intKeys[9] = 6; intValues[9] = 9;

  hipMalloc((void**)&dev_intKeys, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intKeys failed!");

  hipMalloc((void**)&dev_intValues, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intValues failed!");

  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  std::cout << "before unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // How to copy data to the GPU
  hipMemcpy(dev_intKeys, intKeys, sizeof(int) * N, hipMemcpyHostToDevice);
  hipMemcpy(dev_intValues, intValues, sizeof(int) * N, hipMemcpyHostToDevice);

  // Wrap device vectors in thrust iterators for use with thrust.
  thrust::device_ptr<int> dev_thrust_keys(dev_intKeys);
  thrust::device_ptr<int> dev_thrust_values(dev_intValues);
  // LOOK-2.1 Example for using thrust::sort_by_key
  thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + N, dev_thrust_values);

  // How to copy data back to the CPU side from the GPU
  hipMemcpy(intKeys, dev_intKeys, sizeof(int) * N, hipMemcpyDeviceToHost);
  hipMemcpy(intValues, dev_intValues, sizeof(int) * N, hipMemcpyDeviceToHost);
  checkCUDAErrorWithLine("memcpy back failed!");

  std::cout << "after unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // cleanup
  delete(intKeys);
  delete(intValues);
  hipFree(dev_intKeys);
  hipFree(dev_intValues);
  checkCUDAErrorWithLine("hipFree failed!");
  return;
}
