#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <math.h>
#include <glm/glm.hpp>
#include "utilityCore.hpp"
#include "kernel.h"

// LOOK-2.1 potentially useful for doing grid-based neighbor search
#ifndef imax
#define imax( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef imin
#define imin( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)
#define SIM_TIME 0

/**
* Check for CUDA errors; print and exit if there was a problem.
*/
void checkCUDAError(const char *msg, int line = -1) {
	hipError_t err = hipGetLastError();
	if (hipSuccess != err) {
		if (line >= 0) {
			fprintf(stderr, "Line %d: ", line);
		}
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}


/*****************
* Configuration *
*****************/

/*! Block size used for CUDA kernel launch. */
#define blockSize 128

// LOOK-1.2 Parameters for the boids algorithm.
// These worked well in our reference implementation.
#define rule1Distance 5.0f
#define rule2Distance 3.0f
#define rule3Distance 5.0f

#define rule1Scale 0.01f
#define rule2Scale 0.1f
#define rule3Scale 0.1f

#define maxSpeed 1.0f

/*! Size of the starting area in simulation space. */
#define scene_scale 100.0f

#define PROFILE_STEPS 5000
/***********************************************
* Kernel state (pointers are device pointers) *
***********************************************/

int numObjects;
dim3 threadsPerBlock(blockSize);

// LOOK-1.2 - These buffers are here to hold all your boid information.
// These get allocated for you in Boids::initSimulation.
// Consider why you would need two velocity buffers in a simulation where each
// boid cares about its neighbors' velocities.
// These are called ping-pong buffers.
glm::vec3 *dev_pos;
glm::vec3 *dev_vel1;
glm::vec3 *dev_vel2;

// LOOK-2.1 - these are NOT allocated for you. You'll have to set up the thrust
// pointers on your own too.
// For efficient sorting and the uniform grid. These should always be parallel.
int *dev_particleArrayIndices; // What index in dev_pos and dev_velX represents this particle?
int *dev_particleGridIndices; // What grid cell is this particle in?
// needed for use with thrust
thrust::device_ptr<int> dev_thrust_particleArrayIndices;
thrust::device_ptr<int> dev_thrust_particleGridIndices;

int *dev_gridCellStartIndices; // What part of dev_particleArrayIndices belongs
int *dev_gridCellEndIndices;   // to this cell?

// TODO-2.3 - consider what additional buffers you might need to reshuffle
// the position and velocity data to be coherent within cells.
glm::vec3 *dev_rearrangedPos;

// LOOK-2.1 - Grid parameters based on simulation parameters.
// These are automatically computed for you in Boids::initSimulation
int gridCellCount;
int gridSideCount;
float gridCellWidth;
float gridInverseCellWidth;
glm::vec3 gridMinimum;

// Profiling variables
#if SIM_TIME
hipEvent_t start, stop;
double totaltime;
long steps;
#endif

/******************
* initSimulation *
******************/

__host__ __device__ unsigned int hash(unsigned int a) {
	a = (a + 0x7ed55d16) + (a << 12);
	a = (a ^ 0xc761c23c) ^ (a >> 19);
	a = (a + 0x165667b1) + (a << 5);
	a = (a + 0xd3a2646c) ^ (a << 9);
	a = (a + 0xfd7046c5) + (a << 3);
	a = (a ^ 0xb55a4f09) ^ (a >> 16);
	return a;
}

/**
* LOOK-1.2 - this is a typical helper function for a CUDA kernel.
* Function for generating a random vec3.
*/
__host__ __device__ glm::vec3 generateRandomVec3(float time, int index) {
	thrust::default_random_engine rng(hash((int)(index * time)));
	thrust::uniform_real_distribution<float> unitDistrib(-1, 1);

	return glm::vec3((float)unitDistrib(rng), (float)unitDistrib(rng), (float)unitDistrib(rng));
}

/**
* LOOK-1.2 - This is a basic CUDA kernel.
* CUDA kernel for generating boids with a specified mass randomly around the star.
*/
__global__ void kernGenerateRandomPosArray(int time, int N, glm::vec3 * arr, float scale) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index < N) {
		glm::vec3 rand = generateRandomVec3(time, index);
		arr[index].x = scale * rand.x;
		arr[index].y = scale * rand.y;
		arr[index].z = scale * rand.z;
	}
}

/**
* Initialize memory, update some globals
*/
void Boids::initSimulation(int N) {
	numObjects = N;
	dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

	// LOOK-1.2 - This is basic CUDA memory management and error checking.
	// Don't forget to hipFree in  Boids::endSimulation.
	hipMalloc((void**)&dev_pos, N * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMalloc dev_pos failed!");

	hipMalloc((void**)&dev_vel1, N * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMalloc dev_vel1 failed!");

	hipMalloc((void**)&dev_vel2, N * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMalloc dev_vel2 failed!");

	// LOOK-1.2 - This is a typical CUDA kernel invocation.
	kernGenerateRandomPosArray << <fullBlocksPerGrid, blockSize >> >(1, numObjects,
		dev_pos, scene_scale);
	checkCUDAErrorWithLine("kernGenerateRandomPosArray failed!");

	// LOOK-2.1 computing grid params
	gridCellWidth = 2.0f * std::max(std::max(rule1Distance, rule2Distance), rule3Distance);
	int halfSideCount = (int)(scene_scale / gridCellWidth) + 1;
	gridSideCount = 2 * halfSideCount;

	gridCellCount = gridSideCount * gridSideCount * gridSideCount;
	gridInverseCellWidth = 1.0f / gridCellWidth;
	float halfGridWidth = gridCellWidth * halfSideCount;
	gridMinimum.x -= halfGridWidth;
	gridMinimum.y -= halfGridWidth;
	gridMinimum.z -= halfGridWidth;

	// TODO-2.1 TODO-2.3 - Allocate additional buffers here.
	hipMalloc((void**)&dev_particleArrayIndices, N * sizeof(int));
	checkCUDAErrorWithLine("hipMalloc dev_particleArrayIndices failed!");

	hipMalloc((void**)&dev_particleGridIndices, N * sizeof(int));
	checkCUDAErrorWithLine("hipMalloc dev_particleGridIndices failed!");

	hipMalloc((void**)&dev_gridCellStartIndices, gridCellCount * sizeof(int));
	checkCUDAErrorWithLine("hipMalloc dev_gridCellStartIndices failed!");

	hipMalloc((void**)&dev_gridCellEndIndices, gridCellCount * sizeof(int));
	checkCUDAErrorWithLine("hipMalloc dev_gridCellEndIndices failed!");

	hipMalloc((void**)&dev_rearrangedPos, N * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMalloc dev_rearrangedPos failed!");

	dev_thrust_particleArrayIndices = thrust::device_ptr<int>(dev_particleArrayIndices);
	dev_thrust_particleGridIndices = thrust::device_ptr<int>(dev_particleGridIndices);

	#if SIM_TIME
	hipEventCreate(&start);
	hipEventCreate(&stop);
	steps = 0;
	totaltime = 0;
	#endif

	hipDeviceSynchronize();
}


/******************
* copyBoidsToVBO *
******************/

/**
* Copy the boid positions into the VBO so that they can be drawn by OpenGL.
*/
__global__ void kernCopyPositionsToVBO(int N, glm::vec3 *pos, float *vbo, float s_scale) {
	int index = threadIdx.x + (blockIdx.x * blockDim.x);

	float c_scale = -1.0f / s_scale;

	if (index < N) {
		vbo[4 * index + 0] = pos[index].x * c_scale;
		vbo[4 * index + 1] = pos[index].y * c_scale;
		vbo[4 * index + 2] = pos[index].z * c_scale;
		vbo[4 * index + 3] = 1.0f;
	}
}

__global__ void kernCopyVelocitiesToVBO(int N, glm::vec3 *vel, float *vbo, float s_scale) {
	int index = threadIdx.x + (blockIdx.x * blockDim.x);

	if (index < N) {
		vbo[4 * index + 0] = vel[index].x + 0.3f;
		vbo[4 * index + 1] = vel[index].y + 0.3f;
		vbo[4 * index + 2] = vel[index].z + 0.3f;
		vbo[4 * index + 3] = 1.0f;
	}
}

/**
* Wrapper for call to the kernCopyboidsToVBO CUDA kernel.
*/
void Boids::copyBoidsToVBO(float *vbodptr_positions, float *vbodptr_velocities) {
	dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

	kernCopyPositionsToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_pos, vbodptr_positions, scene_scale);
	kernCopyVelocitiesToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_vel1, vbodptr_velocities, scene_scale);

	checkCUDAErrorWithLine("copyBoidsToVBO failed!");

	hipDeviceSynchronize();
}


/******************
* stepSimulation *
******************/

/**
* LOOK-1.2 You can use this as a helper for kernUpdateVelocityBruteForce.
* __device__ code can be called from a __global__ context
* Compute the new velocity on the body with index `iSelf` due to the `N` boids
* in the `pos` and `vel` arrays.
*/
__device__ glm::vec3 computeVelocityChange(int N, int iSelf, const glm::vec3 *pos, const glm::vec3 *vel) {
	glm::vec3 return_vel = vel[iSelf];
	glm::vec3 center = glm::vec3(0.0f, 0.0f, 0.0f);
	glm::vec3 separate = glm::vec3(0.0f, 0.0f, 0.0f);
	glm::vec3 cohesion = glm::vec3(0.0f, 0.0f, 0.0f);
	int neighborCount1 = 0;
	int neighborCount3 = 0;

	for (int j = 0; j < N; j++) {
		if (iSelf == j) continue;
		float distance = glm::distance(pos[iSelf], pos[j]);
		if (distance < rule1Distance) {
			// Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
			center += pos[j];
			neighborCount1 += 1;
		}

		// Rule 2: boids try to stay a distance d away from each other
		if (distance < rule2Distance) {
			separate -= (pos[j] - pos[iSelf]);
		}

		if (distance < rule3Distance) {
			// Rule 3: boids try to match the speed of surrounding boids
			cohesion += vel[j];
			neighborCount3 += 1;
		}
	}
	if (neighborCount1) {
		center /= neighborCount1;
		return_vel += (center - pos[iSelf]) * rule1Scale;
	}
	if (neighborCount3) {
		cohesion /= neighborCount3;
		return_vel += (cohesion - vel[iSelf]) * rule3Scale;
	}
	return_vel += separate * rule2Scale;
	return return_vel;
}

/**
* TODO-1.2 implement basic flocking
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdateVelocityBruteForce(int N, glm::vec3 *pos,
	glm::vec3 *vel1, glm::vec3 *vel2) {
	
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= N) {
		return;
	}
	// Compute a new velocity based on pos and vel1
	glm::vec3 thisBoid_vel2 = computeVelocityChange(N, index, pos, vel1);

	// Clamp the speed
	float speed = glm::length(thisBoid_vel2);
	if (speed > maxSpeed) {
		thisBoid_vel2 = glm::normalize(thisBoid_vel2) * maxSpeed;
	}

	// Record the new velocity into vel2. Question: why NOT vel1?
	vel2[index] = thisBoid_vel2;
}

/**
* LOOK-1.2 Since this is pretty trivial, we implemented it for you.
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdatePos(int N, float dt, glm::vec3 *pos, glm::vec3 *vel) {
	// Update position by velocity
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= N) {
		return;
	}
	glm::vec3 thisPos = pos[index];
	thisPos += vel[index] * dt;

	// Wrap the boids around so we don't lose them
	thisPos.x = thisPos.x < -scene_scale ? scene_scale : thisPos.x;
	thisPos.y = thisPos.y < -scene_scale ? scene_scale : thisPos.y;
	thisPos.z = thisPos.z < -scene_scale ? scene_scale : thisPos.z;

	thisPos.x = thisPos.x > scene_scale ? -scene_scale : thisPos.x;
	thisPos.y = thisPos.y > scene_scale ? -scene_scale : thisPos.y;
	thisPos.z = thisPos.z > scene_scale ? -scene_scale : thisPos.z;

	pos[index] = thisPos;
}

// LOOK-2.1 Consider this method of computing a 1D index from a 3D grid index.
// LOOK-2.3 Looking at this method, what would be the most memory efficient
//          order for iterating over neighboring grid cells?
//          for(x)
//            for(y)
//             for(z)? Or some other order?
__device__ int gridIndex3Dto1D(int x, int y, int z, int gridResolution) {
	if (x < 0 || y < 0 || z < 0 || x > gridResolution || y > gridResolution || z > gridResolution)
		return -1;
	else
		return x + y * gridResolution + z * gridResolution * gridResolution;
}

__global__ void kernComputeIndices(int N, int gridResolution,
	glm::vec3 gridMin, float inverseCellWidth,
	glm::vec3 *pos, int *indices, int *gridIndices) {
	// TODO-2.1
	// - Label each boid with the index of its grid cell.
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= N) {
		return;
	}
	int x = (int)((pos[index].x - gridMin.x) * inverseCellWidth);
	int y = (int)((pos[index].y - gridMin.y) * inverseCellWidth);
	int z = (int)((pos[index].z - gridMin.z) * inverseCellWidth);
	gridIndices[index] = gridIndex3Dto1D(x, y, z, gridResolution);

	// - Set up a parallel array of integer indices as pointers to the actual
	//   boid data in pos and vel1/vel2
	indices[index] = index;
}

// LOOK-2.1 Consider how this could be useful for indicating that a cell
//          does not enclose any boids
__global__ void kernResetIntBuffer(int N, int *intBuffer, int value) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index < N) {
		intBuffer[index] = value;
	}
}

__global__ void kernIdentifyCellStartEnd(int N, int *particleGridIndices,
	int *gridCellStartIndices, int *gridCellEndIndices) {
	// TODO-2.1
	// Identify the start point of each cell in the gridIndices array.
	// This is basically a parallel unrolling of a loop that goes
	// "this index doesn't match the one before it, must be a new cell!"
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index >= N)
		return;
	if ((particleGridIndices[index] != particleGridIndices[index - 1]) || index == 0)
	{
		gridCellStartIndices[particleGridIndices[index]] = index;
	}

	if ((particleGridIndices[index] != particleGridIndices[index + 1]) || index == N - 1)
	{
		gridCellEndIndices[particleGridIndices[index]] = index;
	}
}

__global__ void kernUpdateVelNeighborSearchScattered(
	int N, int gridResolution, glm::vec3 gridMin,
	float inverseCellWidth, float cellWidth,
	int *gridCellStartIndices, int *gridCellEndIndices,
	int *particleArrayIndices,
	glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
	// TODO-2.1 - Update a boid's velocity using the uniform grid to reduce
	// the number of boids that need to be checked.
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index >= N)
		return;
	int neighborGrids[8];
	int startindex;
	int endindex;
	// - Identify the grid cell that this particle is in
	int thisBoid = particleArrayIndices[index];
	// - Identify which cells may contain neighbors. This isn't always 8.
	//findNeighbors(neighborGrids, gridResolution, inverseCellWidth, pos[thisBoid], cellWidth, gridMin);

	float x = ((pos[thisBoid].x - gridMin.x) * inverseCellWidth);
	float y = ((pos[thisBoid].y - gridMin.y) * inverseCellWidth);
	float z = ((pos[thisBoid].z - gridMin.z) * inverseCellWidth);
	int boidx = int(x);
	int boidy = int(y);
	int boidz = int(z);
	int offsetx = (x - boidx) >= 0.5f ? 1 : -1;
	int offsety = (y - boidy) >= 0.5f ? 1 : -1;
	int offsetz = (z - boidz) >= 0.5f ? 1 : -1;
	neighborGrids[0] = gridIndex3Dto1D(boidx,			boidy,				boidz,			 gridResolution);
	neighborGrids[1] = gridIndex3Dto1D(boidx + offsetx, boidy,				boidz,		     gridResolution);
	neighborGrids[2] = gridIndex3Dto1D(boidx,			boidy + offsety,	boidz,			 gridResolution);
	neighborGrids[3] = gridIndex3Dto1D(boidx,			boidy,				boidz + offsetz, gridResolution);
	neighborGrids[4] = gridIndex3Dto1D(boidx + offsetx, boidy + offsety,	boidz,			 gridResolution);
	neighborGrids[5] = gridIndex3Dto1D(boidx,			boidy + offsety,	boidz + offsetz, gridResolution);
	neighborGrids[6] = gridIndex3Dto1D(boidx + offsetx, boidy,				boidz + offsetz, gridResolution);
	neighborGrids[7] = gridIndex3Dto1D(boidx + offsetx, boidy + offsety,	boidz + offsetz, gridResolution);

	glm::vec3 return_vel = vel1[thisBoid];
	glm::vec3 center = glm::vec3(0.0f, 0.0f, 0.0f);
	glm::vec3 separate = glm::vec3(0.0f, 0.0f, 0.0f);
	glm::vec3 cohesion = glm::vec3(0.0f, 0.0f, 0.0f);
	int neighborCount1 = 0;
	int neighborCount3 = 0;

	for (int i = 0; i < 8; i++) {
		if (neighborGrids[i] == -1)
			continue;
		// - For each cell, read the start/end indices in the boid pointer array.
		startindex = gridCellStartIndices[neighborGrids[i]];
		endindex = gridCellStartIndices[neighborGrids[i]];
		// - Access each boid in the cell and compute velocity change from
		//   the boids rules, if this boid is within the neighborhood distance.
		for (int j = startindex; j <= endindex; j++) {
			int thatBoid = particleArrayIndices[j];
			if (thisBoid == thatBoid) continue;
			float distance = glm::distance(pos[thisBoid], pos[thatBoid]);
			if (distance < rule1Distance) {
				// Rule 1: Cohesion: boids fly towards the center of mass of neighboring boids
				center += pos[thatBoid];
				neighborCount1 += 1;
			}

			// Rule 2: Separation: boids try to keep a small distance away from each other
			if (distance < rule2Distance) {
				separate -= (pos[thatBoid] - pos[thisBoid]);
			}

			if (distance < rule3Distance) {
				// Rule 3: Alignment: boids try to match the velocities of neighboring boids
				cohesion += vel1[thatBoid];
				neighborCount3 += 1;
			}
		}
	}
	if (neighborCount1) {
		center /= neighborCount1;
		return_vel += (center - pos[thisBoid]) * rule1Scale;
	}
	if (neighborCount3) {
		cohesion /= neighborCount3;
		return_vel += (cohesion - vel1[thisBoid]) * rule3Scale;
	}
	return_vel += separate * rule2Scale;
	// - Clamp the speed change before putting the new speed in vel2
	//return_vel = glm::clamp(return_vel, glm::vec3(-1.0f, -1.0f, -1.0f), glm::vec3(1.0f, 1.0f, 1.0f));
	float speed = glm::length(return_vel);
	if (speed > maxSpeed) {
		return_vel = glm::normalize(return_vel) * maxSpeed;
	}
	vel2[thisBoid] = return_vel;
}

__global__ void kernUpdateVelNeighborSearchCoherent(
	int N, int gridResolution, glm::vec3 gridMin,
	float inverseCellWidth, float cellWidth,
	int *gridCellStartIndices, int *gridCellEndIndices,
	glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
	// TODO-2.3 - This should be very similar to kernUpdateVelNeighborSearchScattered,
	// except with one less level of indirection.
	// This should expect gridCellStartIndices and gridCellEndIndices to refer
	// directly to pos and vel1.
	// - Identify the grid cell that this particle is in 

	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index >= N)
		return;
	int neighborGrids[8];
	int startindex;
	int endindex;
	// - Identify which cells may contain neighbors. This isn't always 8.
	//findNeighbors(neighborGrids, gridResolution, inverseCellWidth, pos[index], cellWidth, gridMin);

	float x = ((pos[index].x - gridMin.x) * inverseCellWidth);
	float y = ((pos[index].y - gridMin.y) * inverseCellWidth);
	float z = ((pos[index].z - gridMin.z) * inverseCellWidth);
	int boidx = int(x);
	int boidy = int(y);
	int boidz = int(z);
	int offsetx = (x - boidx) >= 0.5f ? 1 : -1;
	int offsety = (y - boidy) >= 0.5f ? 1 : -1;
	int offsetz = (z - boidz) >= 0.5f ? 1 : -1;
	neighborGrids[0] = gridIndex3Dto1D(boidx, boidy, boidz, gridResolution);
	neighborGrids[1] = gridIndex3Dto1D(boidx + offsetx, boidy, boidz, gridResolution);
	neighborGrids[2] = gridIndex3Dto1D(boidx, boidy + offsety, boidz, gridResolution);
	neighborGrids[3] = gridIndex3Dto1D(boidx + offsetx, boidy + offsety, boidz, gridResolution);
	neighborGrids[4] = gridIndex3Dto1D(boidx, boidy, boidz + offsetz, gridResolution);
	neighborGrids[5] = gridIndex3Dto1D(boidx + offsetx, boidy, boidz + offsetz, gridResolution);
	neighborGrids[6] = gridIndex3Dto1D(boidx, boidy + offsety, boidz + offsetz, gridResolution);
	neighborGrids[7] = gridIndex3Dto1D(boidx + offsetx, boidy + offsety, boidz + offsetz, gridResolution);


	// - For each cell, read the start/end indices in the boid pointer array.
	//   DIFFERENCE: For best results, consider what order the cells should be
	//   checked in to maximize the memory benefits of reordering the boids data.
	glm::vec3 return_vel = vel1[index];
	glm::vec3 center = glm::vec3(0.0f, 0.0f, 0.0f);
	glm::vec3 separate = glm::vec3(0.0f, 0.0f, 0.0f);
	glm::vec3 cohesion = glm::vec3(0.0f, 0.0f, 0.0f);
	int neighborCount1 = 0;
	int neighborCount3 = 0;

	for (int i = 0; i < 8; i++) {
		if (neighborGrids[i] == -1)
			continue;

		startindex = gridCellStartIndices[neighborGrids[i]];
		endindex = gridCellStartIndices[neighborGrids[i]];
		// - Access each boid in the cell and compute velocity change from
		//   the boids rules, if this boid is within the neighborhood distance.
		for (int j = startindex; j <= endindex; j++) {
			if (j == index) continue;
			float distance = glm::distance(pos[index], pos[j]);
			if (distance < rule1Distance) {
				// Rule 1: Cohesion: boids fly towards the center of mass of neighboring boids
				center += pos[j];
				neighborCount1 += 1;
			}

			// Rule 2: Separation: boids try to keep a small distance away from each other
			if (distance < rule2Distance) {
				separate -= (pos[j] - pos[index]);
			}

			if (distance < rule3Distance) {
				// Rule 3: Alignment: boids try to match the velocities of neighboring boids
				cohesion += vel1[j];
				neighborCount3 += 1;
			}
		}
	}
	if (neighborCount1) {
		center /= neighborCount1;
		return_vel += (center - pos[index]) * rule1Scale;
	}
	if (neighborCount3) {
		cohesion /= neighborCount3;
		return_vel += (cohesion - vel1[index]) * rule3Scale;
	}
	return_vel += separate * rule2Scale;
	// - Clamp the speed change before putting the new speed in vel2
	float speed = glm::length(return_vel);
	if (speed > maxSpeed) {
		return_vel = glm::normalize(return_vel) * maxSpeed;
	}
	vel2[index] = return_vel;
}

/**
* Kernel function to rearrange the position and velocity vector
* according to the particle array indices.
**/
__global__ void kernelRearrange(int N, int *particleArrayIndices,
	glm::vec3 *orriginalArray, glm::vec3 *rearrangedArray) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index >= N)
		return;
	rearrangedArray[index] = orriginalArray[particleArrayIndices[index]];
}

/**
* Step the entire N-body simulation by `dt` seconds.
*/
void Boids::stepSimulationNaive(float dt) {
	// TODO-1.2 - use the kernels you wrote to step the simulation forward in time.
	dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

	#if SIM_TIME
	hipEventRecord(start);
	#endif

	kernUpdateVelocityBruteForce << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_pos, dev_vel1, dev_vel2);
	hipDeviceSynchronize();
	kernUpdatePos << <fullBlocksPerGrid, blockSize >> >(numObjects, dt, dev_pos, dev_vel2);

	// TODO-1.2 ping-pong the velocity buffers
	glm::vec3 *tmp = dev_vel1;
	dev_vel1 = dev_vel2;
	dev_vel2 = tmp;

	#if SIM_TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	totaltime += milliseconds;
	steps++;
	if (steps == PROFILE_STEPS) {
		printf("Naive average execution time: %f\n", totaltime / steps);
		exit(0);
	}
	#endif
}

void Boids::stepSimulationScatteredGrid(float dt) {
	// TODO-2.1
	// Uniform Grid Neighbor search using Thrust sort.
	dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
	dim3 GridBlocksPerGrid((gridCellCount + blockSize - 1) / blockSize);

	#if SIM_TIME
	hipEventRecord(start);
	#endif

	// In Parallel:
	kernResetIntBuffer << <GridBlocksPerGrid, blockSize >> > (gridCellCount, dev_gridCellStartIndices, -1);
	kernResetIntBuffer << <GridBlocksPerGrid, blockSize >> > (gridCellCount, dev_gridCellEndIndices, -1);

	// - label each particle with its array index as well as its grid index.
	//   Use 2x width grids.
	kernComputeIndices << <fullBlocksPerGrid, blockSize >> > (numObjects, gridSideCount,
		gridMinimum, gridInverseCellWidth, dev_pos, dev_particleArrayIndices, dev_particleGridIndices);
	hipDeviceSynchronize();

	// - Unstable key sort using Thrust. A stable sort isn't necessary, but you
	//   are welcome to do a performance comparison.
	thrust::sort_by_key(dev_thrust_particleGridIndices, dev_thrust_particleGridIndices + numObjects, dev_thrust_particleArrayIndices);

	// - Naively unroll the loop for finding the start and end indices of each
	//   cell's data pointers in the array of boid indices
	kernIdentifyCellStartEnd << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_particleGridIndices,
		dev_gridCellStartIndices, dev_gridCellEndIndices);
	hipDeviceSynchronize();

	// - Perform velocity updates using neighbor search
	kernUpdateVelNeighborSearchScattered << <fullBlocksPerGrid, blockSize >> > (numObjects, gridSideCount, gridMinimum,
		gridInverseCellWidth, gridCellWidth, dev_gridCellStartIndices, dev_gridCellEndIndices,
		dev_particleArrayIndices, dev_pos, dev_vel1, dev_vel2);
	hipDeviceSynchronize();

	// - Update positions
	kernUpdatePos << <fullBlocksPerGrid, blockSize >> >(numObjects, dt, dev_pos, dev_vel2);
	hipDeviceSynchronize();

	// - Ping-pong buffers as needed
	glm::vec3 *tmp = dev_vel1;
	dev_vel1 = dev_vel2;
	dev_vel2 = tmp;

	#if SIM_TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	totaltime += milliseconds;
	steps++;
	if (steps == PROFILE_STEPS) {
		printf("Scattered search execution time: %f\n", totaltime / steps);
		exit(0);
	}
	#endif
}

void Boids::stepSimulationCoherentGrid(float dt) {
	// TODO-2.3 - start by copying Boids::stepSimulationNaiveGrid
	// Uniform Grid Neighbor search using Thrust sort on cell-coherent data.
	dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
	dim3 GridBlocksPerGrid((gridCellCount + blockSize - 1) / blockSize);

	#if SIM_TIME
	hipEventRecord(start);
	#endif

	// In Parallel:
	kernResetIntBuffer << <GridBlocksPerGrid, blockSize >> > (gridCellCount, dev_gridCellStartIndices, -1);
	kernResetIntBuffer << <GridBlocksPerGrid, blockSize >> > (gridCellCount, dev_gridCellEndIndices, -1);

	// - Label each particle with its array index as well as its grid index.
	//   Use 2x width grids
	kernComputeIndices << <fullBlocksPerGrid, blockSize >> > (numObjects, gridSideCount,
		gridMinimum, gridInverseCellWidth, dev_pos, dev_particleArrayIndices, dev_particleGridIndices);
	hipDeviceSynchronize();

	// - Unstable key sort using Thrust. A stable sort isn't necessary, but you
	//   are welcome to do a performance comparison.
	thrust::sort_by_key(dev_thrust_particleGridIndices, dev_thrust_particleGridIndices + numObjects, dev_thrust_particleArrayIndices);

	// - Naively unroll the loop for finding the start and end indices of each
	//   cell's data pointers in the array of boid indices
	kernIdentifyCellStartEnd << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_particleGridIndices,
		dev_gridCellStartIndices, dev_gridCellEndIndices);
	hipDeviceSynchronize();

	// - BIG DIFFERENCE: use the rearranged array index buffer to reshuffle all
	//   the particle data in the simulation array.
	//   CONSIDER WHAT ADDITIONAL BUFFERS YOU NEED
	kernelRearrange << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_particleArrayIndices, dev_pos, dev_rearrangedPos);
	kernelRearrange << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_particleArrayIndices, dev_vel1, dev_vel2);
	hipDeviceSynchronize();

	// - Perform velocity updates using neighbor search
	kernUpdateVelNeighborSearchCoherent << <fullBlocksPerGrid, blockSize >> > (numObjects, gridSideCount, gridMinimum,
		gridInverseCellWidth, gridCellWidth, dev_gridCellStartIndices, dev_gridCellEndIndices,
		dev_rearrangedPos, dev_vel2, dev_vel1);
	hipDeviceSynchronize();

	// - Update positions
	kernUpdatePos << <fullBlocksPerGrid, blockSize >> >(numObjects, dt, dev_rearrangedPos, dev_vel1);
	hipDeviceSynchronize();

	// - Ping-pong buffers as needed. THIS MAY BE DIFFERENT FROM BEFORE.
	glm::vec3 *tmp = dev_pos;
	dev_pos = dev_rearrangedPos;
	dev_rearrangedPos = tmp;	

	#if SIM_TIME
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	totaltime += milliseconds;
	steps++;
	if (steps == PROFILE_STEPS) {
		printf("Coherent search execution time: %f\n", totaltime / steps);
		exit(0);
	}
	#endif
}

void Boids::endSimulation() {
	hipFree(dev_vel1);
	hipFree(dev_vel2);
	hipFree(dev_pos);

	// TODO-2.1 TODO-2.3 - Free any additional buffers here.
	hipFree(dev_particleArrayIndices);
	hipFree(dev_particleGridIndices);
	hipFree(dev_gridCellStartIndices);
	hipFree(dev_gridCellEndIndices);
	hipFree(dev_rearrangedPos);

	#if SIM_TIME
	hipEventDestroy(start);
	hipEventDestroy(stop);
	#endif
}

void Boids::unitTest() {
	// LOOK-1.2 Feel free to write additional tests here.

	// test unstable sort
	int *dev_intKeys;
	int *dev_intValues;
	int N = 10;

	int *intKeys = new int[N];
	int *intValues = new int[N];

	intKeys[0] = 0; intValues[0] = 0;
	intKeys[1] = 1; intValues[1] = 1;
	intKeys[2] = 0; intValues[2] = 2;
	intKeys[3] = 3; intValues[3] = 3;
	intKeys[4] = 0; intValues[4] = 4;
	intKeys[5] = 2; intValues[5] = 5;
	intKeys[6] = 2; intValues[6] = 6;
	intKeys[7] = 0; intValues[7] = 7;
	intKeys[8] = 5; intValues[8] = 8;
	intKeys[9] = 6; intValues[9] = 9;

	hipMalloc((void**)&dev_intKeys, N * sizeof(int));
	checkCUDAErrorWithLine("hipMalloc dev_intKeys failed!");

	hipMalloc((void**)&dev_intValues, N * sizeof(int));
	checkCUDAErrorWithLine("hipMalloc dev_intValues failed!");

	dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

	/*std::cout << "before unstable sort: " << std::endl;
	for (int i = 0; i < N; i++) {
		std::cout << "  key: " << intKeys[i];
		std::cout << " value: " << intValues[i] << std::endl;
	}*/

	// How to copy data to the GPU
	hipMemcpy(dev_intKeys, intKeys, sizeof(int)* N, hipMemcpyHostToDevice);
	hipMemcpy(dev_intValues, intValues, sizeof(int)* N, hipMemcpyHostToDevice);

	// Wrap device vectors in thrust iterators for use with thrust.
	thrust::device_ptr<int> dev_thrust_keys(dev_intKeys);
	thrust::device_ptr<int> dev_thrust_values(dev_intValues);
	// LOOK-2.1 Example for using thrust::sort_by_key
	thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + N, dev_thrust_values);

	// How to copy data back to the CPU side from the GPU
	hipMemcpy(intKeys, dev_intKeys, sizeof(int)* N, hipMemcpyDeviceToHost);
	hipMemcpy(intValues, dev_intValues, sizeof(int)* N, hipMemcpyDeviceToHost);
	checkCUDAErrorWithLine("memcpy back failed!");

	/*std::cout << "after unstable sort: " << std::endl;
	for (int i = 0; i < N; i++) {
		std::cout << "  key: " << intKeys[i];
		std::cout << " value: " << intValues[i] << std::endl;
	}*/

	// cleanup
	delete[] intKeys;
	delete[] intValues;
	hipFree(dev_intKeys);
	hipFree(dev_intValues);
	checkCUDAErrorWithLine("hipFree failed!");
	return;
}
