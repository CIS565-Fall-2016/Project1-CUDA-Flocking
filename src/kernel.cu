#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <glm/glm.hpp>
#include "utilityCore.hpp"
#include "kernel.h"

#define INVALID_VALUE (-1)

// LOOK-2.1 potentially useful for doing grid-based neighbor search
#ifndef imax
#define imax( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef imin
#define imin( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

/**
* Check for CUDA errors; print and exit if there was a problem.
*/
void checkCUDAError(const char *msg, int line = -1) {
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    if (line >= 0) {
      fprintf(stderr, "Line %d: ", line);
    }
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}


/*****************
* Configuration *
*****************/

/*! Block size used for CUDA kernel launch. */
#define blockSize 128

// LOOK-1.2 Parameters for the boids algorithm.
// These worked well in our reference implementation.
#define rule1Distance 5.0f
#define rule2Distance 3.0f
#define rule3Distance 5.0f

#define rule1Scale 0.01f
#define rule2Scale 0.1f
#define rule3Scale 0.1f

#define maxSpeed 1.0f

/*! Size of the starting area in simulation space. */
#define scene_scale 100.0f

/***********************************************
* Kernel state (pointers are device pointers) *
***********************************************/

int numObjects;
dim3 threadsPerBlock(blockSize);

// LOOK-1.2 - These buffers are here to hold all your boid information.
// These get allocated for you in Boids::initSimulation.
// Consider why you would need two velocity buffers in a simulation where each
// boid cares about its neighbors' velocities.
// These are called ping-pong buffers.
glm::vec3 *dev_pos;
glm::vec3 *dev_vel1;
glm::vec3 *dev_vel2;

// LOOK-2.1 - these are NOT allocated for you. You'll have to set up the thrust
// pointers on your own too.

// For efficient sorting and the uniform grid. These should always be parallel.
int *dev_particleArrayIndices; // What index in dev_pos and dev_velX represents this particle?
int *dev_particleGridIndices; // What grid cell is this particle in?
// needed for use with thrust
thrust::device_ptr<int> dev_thrust_particleArrayIndices;
thrust::device_ptr<int> dev_thrust_particleGridIndices;

int *dev_gridCellStartIndices; // What part of dev_particleArrayIndices belongs
int *dev_gridCellEndIndices;   // to this cell?



// TODO-2.3 - consider what additional buffers you might need to reshuffle
// the position and velocity data to be coherent within cells.
glm::vec3* dev_sorted_pos;
glm::vec3* dev_sorted_vel1;
glm::vec3* dev_sorted_vel2;

// LOOK-2.1 - Grid parameters based on simulation parameters.
// These are automatically computed for you in Boids::initSimulation
int gridCellCount;
int gridSideCount;
float gridCellWidth;
float gridInverseCellWidth;
glm::vec3 gridMinimum;

/******************
* initSimulation *
******************/

__host__ __device__ unsigned int hash(unsigned int a) {
  a = (a + 0x7ed55d16) + (a << 12);
  a = (a ^ 0xc761c23c) ^ (a >> 19);
  a = (a + 0x165667b1) + (a << 5);
  a = (a + 0xd3a2646c) ^ (a << 9);
  a = (a + 0xfd7046c5) + (a << 3);
  a = (a ^ 0xb55a4f09) ^ (a >> 16);
  return a;
}

/**
* LOOK-1.2 - this is a typical helper function for a CUDA kernel.
* Function for generating a random vec3.
*/
__host__ __device__ glm::vec3 generateRandomVec3(float time, int index) {
  thrust::default_random_engine rng(hash((int)(index * time)));
  thrust::uniform_real_distribution<float> unitDistrib(-1, 1);

  return glm::vec3((float)unitDistrib(rng), (float)unitDistrib(rng), (float)unitDistrib(rng));
}

/**
* LOOK-1.2 - This is a basic CUDA kernel.
* CUDA kernel for generating boids with a specified mass randomly around the star.
*/
__global__ void kernGenerateRandomPosArray(int time, int N, glm::vec3 * arr, float scale) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    glm::vec3 rand = generateRandomVec3(time, index);
    arr[index].x = scale * rand.x;
    arr[index].y = scale * rand.y;
    arr[index].z = scale * rand.z;
  }
}

__global__ void kernCopyBuffer(int N, glm::vec3* desBuffer, glm::vec3* srcBuffer) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }

  desBuffer[index] = srcBuffer[index];
}

/**
* Initialize memory, update some globals
*/
void Boids::initSimulation(int N) {
  numObjects = N;
  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  // LOOK-1.2 - This is basic CUDA memory management and error checking.
  // Don't forget to hipFree in  Boids::endSimulation.
  hipMalloc((void**)&dev_pos, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_pos failed!");

  hipMalloc((void**)&dev_vel1, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel1 failed!");

  hipMalloc((void**)&dev_vel2, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel2 failed!");

  hipMalloc((void**)&dev_sorted_pos, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_sorted_pos failed!");
  
  hipMalloc((void**)&dev_sorted_vel1, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_sorted_vel1 failed!");
  hipMalloc((void**)&dev_sorted_vel2, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_sorted_vel2 failed!");

  // LOOK-1.2 - This is a typical CUDA kernel invocation.
  kernGenerateRandomPosArray<<<fullBlocksPerGrid, blockSize>>>(1, numObjects,
    dev_pos, scene_scale);
  checkCUDAErrorWithLine("kernGenerateRandomPosArray failed!");

  // Initialize the dev_sorted_pos
  kernCopyBuffer << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_sorted_pos, dev_pos);

  // LOOK-2.1 computing grid params
  gridCellWidth = 2.0f * std::max(std::max(rule1Distance, rule2Distance), rule3Distance);
  int halfSideCount = (int)(scene_scale / gridCellWidth) + 1;
  gridSideCount = halfSideCount * 2;

  gridCellCount = gridSideCount * gridSideCount * gridSideCount;
  gridInverseCellWidth = 1.0f / gridCellWidth;
  float halfGridWidth = gridCellWidth * halfSideCount;
  gridMinimum.x -= halfGridWidth;
  gridMinimum.y -= halfGridWidth;
  gridMinimum.z -= halfGridWidth;

  // TODO-2.1 TODO-2.3 - Allocate additional buffers here.
  hipMalloc((void**)&dev_particleArrayIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleArrayIndices failed!");
  hipMalloc((void**)&dev_particleGridIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleGridIndices failed!");

  dev_thrust_particleArrayIndices = thrust::device_pointer_cast<int>(dev_particleArrayIndices);
  dev_thrust_particleGridIndices = thrust::device_pointer_cast<int>(dev_particleGridIndices);

  hipMalloc((void**)&dev_gridCellStartIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellStartIndices failed!");
  hipMalloc((void**)&dev_gridCellEndIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellEndIndices failed!");


  hipDeviceSynchronize();
}


/******************
* copyBoidsToVBO *
******************/

/**
* Copy the boid positions into the VBO so that they can be drawn by OpenGL.
*/
__global__ void kernCopyPositionsToVBO(int N, glm::vec3 *pos, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  float c_scale = -1.0f / s_scale;

  if (index < N) {
    vbo[4 * index + 0] = pos[index].x * c_scale;
    vbo[4 * index + 1] = pos[index].y * c_scale;
    vbo[4 * index + 2] = pos[index].z * c_scale;
    vbo[4 * index + 3] = 1.0f;
  }
}

__global__ void kernCopyVelocitiesToVBO(int N, glm::vec3 *vel, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  if (index < N) {
    vbo[4 * index + 0] = vel[index].x + 0.3f;
    vbo[4 * index + 1] = vel[index].y + 0.3f;
    vbo[4 * index + 2] = vel[index].z + 0.3f;
    vbo[4 * index + 3] = 1.0f;
  }
}

/**
* Wrapper for call to the kernCopyboidsToVBO CUDA kernel.
*/
void Boids::copyBoidsToVBO(float *vbodptr_positions, float *vbodptr_velocities) {
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

  kernCopyPositionsToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_pos, vbodptr_positions, scene_scale);
  kernCopyVelocitiesToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_vel1, vbodptr_velocities, scene_scale);

  checkCUDAErrorWithLine("copyBoidsToVBO failed!");

  hipDeviceSynchronize();
}

/******************
* stepSimulation *
******************/

/**
* LOOK-1.2 You can use this as a helper for kernUpdateVelocityBruteForce.
* __device__ code can be called from a __global__ context
* Compute the new velocity on the body with index `iSelf` due to the `N` boids
* in the `pos` and `vel` arrays.
*/
__device__ glm::vec3 computeVelocityChange(int N, int iSelf, const glm::vec3 *pos, const glm::vec3 *vel) {
  
  glm::vec3 velCohesion, velSeparation, velAlignment;
  glm::vec3 centerOfMass;
  unsigned int rule1NeighborCount = 0;
  unsigned int rule2NeighborCount = 0;
  unsigned int rule3NeighborCount = 0;
  
  for (auto i = 0; i < N; ++i) {
    
    // skip itself
    if (i == iSelf) continue;
    auto distance = glm::distance(pos[iSelf], pos[i]);

    // Rule 1, cohesion: boids fly towards their local perceived center of mass, which excludes themselves
    if (distance < rule1Distance) {
      centerOfMass += pos[i];
      ++rule1NeighborCount;
    }

    // Rule 2: boids try to stay a distance d away from each other
    if (distance < rule2Distance) {
      velSeparation -= (pos[i] - pos[iSelf]);
      ++rule2NeighborCount;
    }

    // Rule 3: boids try to match the speed of surrounding boids
    if (distance < rule3Distance) {
      velAlignment += vel[i];
      ++rule3NeighborCount;
    }
  }

  if (rule1NeighborCount > 0) {
    centerOfMass /= rule1NeighborCount;
    velCohesion = (centerOfMass - pos[iSelf]) * rule1Scale;
  }

  if (rule2NeighborCount > 0) {
    velSeparation *= rule2Scale;
  }

  if (rule3NeighborCount > 0) {
    velAlignment /= rule3NeighborCount;
    velAlignment = velAlignment - vel[iSelf];
    velAlignment *= rule3Scale;
  }

  // Return new computed velocity
  return velCohesion + velSeparation + velAlignment;
}

/**
* TODO-1.2 implement basic flocking
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdateVelocityBruteForce(int N, glm::vec3 *pos,
  glm::vec3 *vel1, glm::vec3 *vel2) {

  // Compute a new velocity based on pos and vel1
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }
    
  // Record the new velocity into vel2. Question: why NOT vel1?
  // Since the neighbor might be inspecting our current vel1 this frame, we don't want to update vel1 yet
  vel2[index] = vel1[index] + computeVelocityChange(N, index, pos, vel1);

  // Clamp the speed
  if (glm::length(vel2[index]) >= maxSpeed) {
    vel2[index] = glm::normalize(vel2[index]) * maxSpeed;
  }
}

/**
* LOOK-1.2 Since this is pretty trivial, we implemented it for you.
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdatePos(int N, float dt, glm::vec3 *pos, glm::vec3 *vel) {
  // Update position by velocity
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }
  glm::vec3 thisPos = pos[index];
  thisPos += vel[index] * dt;

  // Wrap the boids around so we don't lose them
  thisPos.x = thisPos.x < -scene_scale ? scene_scale : thisPos.x;
  thisPos.y = thisPos.y < -scene_scale ? scene_scale : thisPos.y;
  thisPos.z = thisPos.z < -scene_scale ? scene_scale : thisPos.z;

  thisPos.x = thisPos.x > scene_scale ? -scene_scale : thisPos.x;
  thisPos.y = thisPos.y > scene_scale ? -scene_scale : thisPos.y;
  thisPos.z = thisPos.z > scene_scale ? -scene_scale : thisPos.z;

  pos[index] = thisPos;
}

// LOOK-2.1 Consider this method of computing a 1D index from a 3D grid index.
// LOOK-2.3 Looking at this method, what would be the most memory efficient
//          order for iterating over neighboring grid cells?
//          for(x)
//            for(y)
//             for(z)? Or some other order?
__device__ int gridIndex3Dto1D(int x, int y, int z, int gridResolution) {
  return x + y * gridResolution + z * gridResolution * gridResolution;
}

__global__ void kernComputeIndices(int N, int gridResolution,
  glm::vec3 gridMin, float inverseCellWidth,
  glm::vec3 *pos, int *indices, int *gridIndices) {
    // TODO-2.1
    // - Label each boid with the index of its grid cell.
    // - Set up a parallel array of integer indices as pointers to the actual
    //   boid data in pos and vel1/vel2

  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    indices[index] = index;

    // Compute the 3D grid indices for this particle
    int gridX = (int)((pos[index].x - gridMin.x) * inverseCellWidth);
    int gridY = (int)((pos[index].y - gridMin.y) * inverseCellWidth);
    int gridZ = (int)((pos[index].z - gridMin.z) * inverseCellWidth);

    // Convert and store the 1D grid index
    gridIndices[index] = gridIndex3Dto1D(gridX, gridY, gridZ, gridResolution);
  }
}

// LOOK-2.1 Consider how this could be useful for indicating that a cell
//          does not enclose any boids
__global__ void kernResetIntBuffer(int N, int *intBuffer, int value) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    intBuffer[index] = value;
  }
}

__global__ void kernIdentifyCellStartEnd(int N, int *particleGridIndices,
  int *gridCellStartIndices, int *gridCellEndIndices) {
  // TODO-2.1
  // Identify the start point of each cell in the gridIndices array.
  // This is basically a parallel unrolling of a loop that goes
  // "this index doesn't match the one before it, must be a new cell!"

  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index > N) {
    return;
  }

  int cellIndex = particleGridIndices[index];
  if (index == 0) {
    gridCellStartIndices[cellIndex] = 0;
    return;
  }

  if (index == N - 1) {
    gridCellEndIndices[cellIndex] = N - 1;
  }

  int prevCellIndex = particleGridIndices[index - 1];
  if (cellIndex != prevCellIndex)
  {
    gridCellStartIndices[cellIndex] = index;
    gridCellEndIndices[prevCellIndex] = index - 1;
  }
}

__global__ void kernUpdateVelNeighborSearchScattered(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  int *particleArrayIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  // TODO-2.1 - Update a boid's velocity using the uniform grid to reduce
  // the number of boids that need to be checked.
  // - Identify the grid cell that this particle is in
  // - Identify which cells may contain neighbors. This isn't always 8.
  // - For each cell, read the start/end indices in the boid pointer array.
  // - Access each boid in the cell and compute velocity change from
  //   the boids rules, if this boid is within the neighborhood distance.
  // - Clamp the speed change before putting the new speed in vel2

  // Compute a new velocity based on pos and vel1
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }

  int boidIndex = particleArrayIndices[index];

  // Compute the 3D grid indices for this particle
  glm::vec3 grid3DIndex = (pos[boidIndex] - gridMin) * inverseCellWidth;

    // Find the other 8 neiboring cells based on which octant the particle is in
  glm::vec3 distanceToNeighbor = 
    pos[boidIndex] - (glm::vec3((int)grid3DIndex.x, (int)grid3DIndex.y, (int)grid3DIndex.z) * cellWidth + gridMin);
  float halfCellWidth = cellWidth / 2.0f;

  int neighborMinIndexX, neighborMinIndexY, neighborMinIndexZ, neighborMaxIndexX, neighborMaxIndexY, neighborMaxIndexZ;

  if (distanceToNeighbor.x < halfCellWidth) {
    neighborMinIndexX = (int)grid3DIndex.x - 1 >= 0 ? (int)grid3DIndex.x - 1 : (int)grid3DIndex.x;
    neighborMaxIndexX = (int)grid3DIndex.x;
  } else {
    neighborMinIndexX = (int)grid3DIndex.x;
    neighborMaxIndexX = (int)grid3DIndex.x + 1 < N ? (int)grid3DIndex.x + 1: (int)grid3DIndex.x;
  }

  if (distanceToNeighbor.y < halfCellWidth) {
    neighborMinIndexY = (int)grid3DIndex.y - 1 >= 0 ? (int)grid3DIndex.y - 1 : (int)grid3DIndex.y;
    neighborMaxIndexY = (int)grid3DIndex.y;
  }
  else {
    neighborMinIndexY = (int)grid3DIndex.y;
    neighborMaxIndexY = (int)grid3DIndex.y + 1 < N ? (int)grid3DIndex.y + 1 : (int)grid3DIndex.y;
  }
  
  if (distanceToNeighbor.z < halfCellWidth) {
    neighborMinIndexZ = (int)grid3DIndex.z - 1 >= 0 ? (int)grid3DIndex.z - 1 : (int)grid3DIndex.z;
    neighborMaxIndexZ = (int)grid3DIndex.z;
  }
  else {
    neighborMinIndexZ = (int)grid3DIndex.z;
    neighborMaxIndexZ = (int)grid3DIndex.z + 1 < N ? (int)grid3DIndex.z + 1 : (int)grid3DIndex.z;
  }

  // These are to keep track of velocity computation
  glm::vec3 velCohesion, velSeparation, velAlignment;
  glm::vec3 centerOfMass;
  unsigned int rule1NeighborCount = 0;
  unsigned int rule2NeighborCount = 0;
  unsigned int rule3NeighborCount = 0;

  // Loop through neighbor cells and find all boids in them
  int neighborIndex, neighborCellStartIndex, neighborCellEndIndex;
  for (auto z = neighborMinIndexZ; z <= neighborMaxIndexZ; ++z) {
    for (auto y = neighborMinIndexY; y <= neighborMaxIndexY; ++y) {
      for (auto x = neighborMinIndexX; x <= neighborMaxIndexX; ++x) {
        
        // Compute velocity contribution from particles in this neighboring cell
        neighborIndex = gridIndex3Dto1D(x, y, z, gridResolution);
        neighborCellStartIndex = gridCellStartIndices[neighborIndex];
        neighborCellEndIndex = gridCellEndIndices[neighborIndex];

        if (neighborCellStartIndex >= 0 && neighborCellStartIndex <= neighborCellEndIndex &&
          neighborCellEndIndex >= neighborCellStartIndex && neighborCellEndIndex < N) {

          //Compute velocity
          for (auto i = neighborCellStartIndex; i < neighborCellEndIndex; ++i) {

            int neighborBoidIndex = particleArrayIndices[i];
            // skip itself
            if (neighborBoidIndex == boidIndex) continue;
            auto distance = glm::distance(pos[boidIndex], pos[neighborBoidIndex]);

            // Rule 1, cohesion: boids fly towards their local perceived center of mass, which excludes themselves
            if (distance < rule1Distance) {
              centerOfMass += pos[neighborBoidIndex];
              ++rule1NeighborCount;
            }

            // Rule 2: boids try to stay a distance d away from each other
            if (distance < rule2Distance) {
              velSeparation -= (pos[neighborBoidIndex] - pos[boidIndex]);
              ++rule2NeighborCount;
            }

            // Rule 3: boids try to match the speed of surrounding boids
            if (distance < rule3Distance) {
              velAlignment += vel1[neighborBoidIndex];
              ++rule3NeighborCount;
            }
          }          
        }
      }
    }
  }

  if (rule1NeighborCount > 0) {
    centerOfMass /= rule1NeighborCount;
    velCohesion = (centerOfMass - pos[boidIndex]) * rule1Scale;
  }

  if (rule2NeighborCount > 0) {
    velSeparation *= rule2Scale;
  }

  if (rule3NeighborCount > 0) {
    velAlignment /= rule3NeighborCount;
    velAlignment = velAlignment - vel1[boidIndex];
    velAlignment *= rule3Scale;
  }

  // Record the new velocity into vel2. Question: why NOT vel1?
  // Since the neighbor might be inspecting our current vel1 this frame, we don't want to update vel1 yet
  vel2[boidIndex] = vel1[boidIndex] + velCohesion + velSeparation + velAlignment;

  // Clamp the speed
  if (glm::length(vel2[boidIndex]) >= maxSpeed) {
    vel2[boidIndex] = glm::normalize(vel2[boidIndex]) * maxSpeed;
  }
}

__global__ void kernUpdateVelNeighborSearchCoherent(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  // TODO-2.3 - This should be very similar to kernUpdateVelNeighborSearchScattered,
  // except with one less level of indirection.
  // This should expect gridCellStartIndices and gridCellEndIndices to refer
  // directly to pos and vel1.
  // - Identify the grid cell that this particle is in
  // - Identify which cells may contain neighbors. This isn't always 8.
  // - For each cell, read the start/end indices in the boid pointer array.
  //   DIFFERENCE: For best results, consider what order the cells should be
  //   checked in to maximize the memory benefits of reordering the boids data.
  // - Access each boid in the cell and compute velocity change from
  //   the boids rules, if this boid is within the neighborhood distance.
  // - Clamp the speed change before putting the new speed in vel2
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }

  // Compute the 3D grid indices for this particle
  glm::vec3 grid3DIndex = (pos[index] - gridMin) * inverseCellWidth;

  // Find the other 8 neiboring cells based on which octant the particle is in
  glm::vec3 distanceToNeighbor =
    pos[index] - (glm::vec3((int)grid3DIndex.x, (int)grid3DIndex.y, (int)grid3DIndex.z) * cellWidth + gridMin);
  float halfCellWidth = cellWidth / 2.0f;

  int neighborMinIndexX, 
    neighborMinIndexY, 
      neighborMinIndexZ, 
        neighborMaxIndexX, 
          neighborMaxIndexY, 
            neighborMaxIndexZ;

  if (distanceToNeighbor.x < halfCellWidth) {
    neighborMinIndexX = (int)grid3DIndex.x - 1;
    neighborMaxIndexX = (int)grid3DIndex.x;
  }
  else {
    neighborMinIndexX = (int)grid3DIndex.x;
    neighborMaxIndexX = (int)grid3DIndex.x + 1;
  }

  if (distanceToNeighbor.y < halfCellWidth) {
    neighborMinIndexY = (int)grid3DIndex.y - 1;
    neighborMaxIndexY = (int)grid3DIndex.y;
  }
  else {
    neighborMinIndexY = (int)grid3DIndex.y;
    neighborMaxIndexY = (int)grid3DIndex.y + 1;
  }

  if (distanceToNeighbor.z < halfCellWidth) {
    neighborMinIndexZ = (int)grid3DIndex.z - 1;
    neighborMaxIndexZ = (int)grid3DIndex.z;
  }
  else {
    neighborMinIndexZ = (int)grid3DIndex.z;
    neighborMaxIndexZ = (int)grid3DIndex.z + 1;
  }

  // These are to keep track of velocity computation
  glm::vec3 velCohesion, velSeparation, velAlignment;
  glm::vec3 centerOfMass;
  unsigned int rule1NeighborCount = 0;
  unsigned int rule2NeighborCount = 0;
  unsigned int rule3NeighborCount = 0;

  // Loop through neighbor cells and find all boids in them
  int neighborIndex, neighborCellStartIndex, neighborCellEndIndex;
  for (int z = neighborMinIndexZ; z <= neighborMaxIndexZ; ++z) {
    for (int y = neighborMinIndexY; y <= neighborMaxIndexY; ++y) {
      for (int x = neighborMinIndexX; x <= neighborMaxIndexX; ++x) {

        // Compute velocity contribution from particles in this neighboring cell
        neighborIndex = gridIndex3Dto1D(x % gridResolution, y % gridResolution, z % gridResolution, gridResolution);
        neighborCellStartIndex = gridCellStartIndices[neighborIndex];
        neighborCellEndIndex = gridCellEndIndices[neighborIndex];

        if (neighborCellStartIndex >= 0 && neighborCellStartIndex <= neighborCellEndIndex &&
          neighborCellEndIndex >= neighborCellStartIndex && neighborCellEndIndex < N) {


          //Compute velocity
          for (auto neighborBoidIndex = neighborCellStartIndex; neighborBoidIndex < neighborCellEndIndex; ++neighborBoidIndex) {

            // skip itself
            if (neighborBoidIndex == index) continue;
            auto distance = glm::distance(pos[index], pos[neighborBoidIndex]);

            // Rule 1, cohesion: boids fly towards their local perceived center of mass, which excludes themselves
            if (distance < rule1Distance) {
              centerOfMass += pos[neighborBoidIndex];
              ++rule1NeighborCount;
            }

            // Rule 2: boids try to stay a distance d away from each other
            if (distance < rule2Distance) {
              velSeparation -= (pos[neighborBoidIndex] - pos[index]);
              ++rule2NeighborCount;
            }

            // Rule 3: boids try to match the speed of surrounding boids
            if (distance < rule3Distance) {
              velAlignment += vel1[neighborBoidIndex];
              ++rule3NeighborCount;
            }
          }
        }
      }
    }
  }

  if (rule1NeighborCount > 0) {
    centerOfMass /= rule1NeighborCount;
    velCohesion = (centerOfMass - pos[index]) * rule1Scale;
  }

  if (rule2NeighborCount > 0) {
    velSeparation *= rule2Scale;
  }

  if (rule3NeighborCount > 0) {
    velAlignment /= rule3NeighborCount;
    velAlignment = velAlignment - vel1[index];
    velAlignment *= rule3Scale;
  }

  // Record the new velocity into vel2. Question: why NOT vel1?
  // Since the neighbor might be inspecting our current vel1 this frame, we don't want to update vel1 yet
  vel2[index] = vel1[index] + velCohesion + velSeparation + velAlignment;

  // Clamp the speed
  if (glm::length(vel2[index]) >= maxSpeed) {
    vel2[index] = glm::normalize(vel2[index]) * maxSpeed;
  } 
}

__global__ void kernRearrangeBuffer(int N, int* particleGridIndices, glm::vec3* newBuffer, glm::vec3* oldBuffer) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }
  int oldIdx = particleGridIndices[index];
  newBuffer[index] = oldBuffer[oldIdx];
}

__global__ void kernCopyBack(int N, int* particleGridIndices, glm::vec3* newBuffer, glm::vec3* oldBuffer) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }

  int newIdx = particleGridIndices[index];
  newBuffer[newIdx] = oldBuffer[index];
}

/**
* Step the entire N-body simulation by `dt` seconds.
*/
void Boids::stepSimulationNaive(float dt) {
  // TODO-1.2 - use the kernels you wrote to step the simulation forward in time.
  static bool isDevVel1Active = true;
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
  
  kernUpdateVelocityBruteForce << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_pos, isDevVel1Active ? dev_vel1 : dev_vel2, isDevVel1Active ? dev_vel2 : dev_vel1);
  kernUpdatePos << <fullBlocksPerGrid, blockSize >> >(numObjects, dt, dev_pos, isDevVel1Active ? dev_vel2 : dev_vel1);

  // TODO-1.2 ping-pong the velocity buffers
  isDevVel1Active = !isDevVel1Active;
}

void Boids::stepSimulationScatteredGrid(float dt) {
  // TODO-2.1
  // Uniform Grid Neighbor search using Thrust sort.
  // In Parallel:
  // - label each particle with its array index as well as its grid index.
  //   Use 2x width grids.
  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
  // - Perform velocity updates using neighbor search
  // - Update positions
  // - Ping-pong buffers as needed

  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

  // reseet the grid cell start and end indices to an invalid value
  kernResetIntBuffer << <fullBlocksPerGrid, blockSize >> >(
    numObjects, 
      dev_gridCellStartIndices, 
      INVALID_VALUE
          );

  kernResetIntBuffer << <fullBlocksPerGrid, blockSize >> >(
    numObjects, 
      dev_gridCellEndIndices, 
      INVALID_VALUE
          );

  // Compute the new index
  kernComputeIndices << <fullBlocksPerGrid, blockSize >> >(
    numObjects, 
      gridSideCount, 
        gridMinimum, 
          gridInverseCellWidth, 
          dev_pos,
              dev_particleArrayIndices, 
                dev_particleGridIndices
                  );

  thrust::sort_by_key(
    dev_thrust_particleGridIndices, 
      dev_thrust_particleGridIndices + numObjects, 
        dev_thrust_particleArrayIndices);

  kernIdentifyCellStartEnd << <fullBlocksPerGrid, blockSize >> >(
    numObjects, 
      dev_particleGridIndices, 
        dev_gridCellStartIndices, 
          dev_gridCellEndIndices
            );

  // Update velocity
  static bool isDevVel1Active = true;
  kernUpdateVelNeighborSearchScattered << <fullBlocksPerGrid, blockSize >> >(
    numObjects,
      gridSideCount, 
        gridMinimum, 
          gridInverseCellWidth, 
            gridCellWidth, 
              dev_gridCellStartIndices, 
                dev_gridCellEndIndices, 
                  dev_particleArrayIndices, 
                    dev_pos, 
                      isDevVel1Active ? dev_vel1 : dev_vel2, 
                        isDevVel1Active ? dev_vel2 : dev_vel1
                          );
  
  //  Update position
  kernUpdatePos << <fullBlocksPerGrid, blockSize >> >(
    numObjects, 
      dt, 
        dev_pos, 
          isDevVel1Active ? dev_vel2 : dev_vel1
            );
  
  // Ping pong
  isDevVel1Active = !isDevVel1Active;
}

void Boids::stepSimulationCoherentGrid(float dt) {
  // TODO-2.3 - start by copying Boids::stepSimulationNaiveGrid
  // Uniform Grid Neighbor search using Thrust sort on cell-coherent data.
  // In Parallel:
  // - Label each particle with its array index as well as its grid index.
  //   Use 2x width grids
  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
  // - BIG DIFFERENCE: use the rearranged array index buffer to reshuffle all
  //   the particle data in the simulation array.
  //   CONSIDER WHAT ADDITIONAL BUFFERS YOU NEED
  // - Perform velocity updates using neighbor search
  // - Update positions
  // - Ping-pong buffers as needed. THIS MAY BE DIFFERENT FROM BEFORE.

  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

  // reseet the grid cell start and end indices to an invalid value
  kernResetIntBuffer << <fullBlocksPerGrid, blockSize >> >(
    numObjects,
    dev_gridCellStartIndices,
    INVALID_VALUE
    );

  kernResetIntBuffer << <fullBlocksPerGrid, blockSize >> >(
    numObjects,
    dev_gridCellEndIndices,
    INVALID_VALUE
    );

  // Compute the new index
  kernComputeIndices << <fullBlocksPerGrid, blockSize >> >(
    numObjects,
    gridSideCount,
    gridMinimum,
    gridInverseCellWidth,
    dev_pos,
    dev_particleArrayIndices,
    dev_particleGridIndices
    );

  thrust::sort_by_key(
    dev_thrust_particleGridIndices,
    dev_thrust_particleGridIndices + numObjects,
    dev_thrust_particleArrayIndices);

  kernIdentifyCellStartEnd << <fullBlocksPerGrid, blockSize >> >(
    numObjects,
    dev_particleGridIndices,
    dev_gridCellStartIndices,
    dev_gridCellEndIndices
    );

  kernRearrangeBuffer << <fullBlocksPerGrid, blockSize >> >(
    numObjects,
    dev_particleArrayIndices,
    dev_sorted_pos,
    dev_pos
    );

  kernRearrangeBuffer << <fullBlocksPerGrid, blockSize >> >(
    numObjects,
    dev_particleArrayIndices,
    dev_vel1,
    dev_vel2
    );

  // Update velocity
  kernUpdateVelNeighborSearchCoherent << <fullBlocksPerGrid, blockSize >> >(
    numObjects,
    gridSideCount,
    gridMinimum,
    gridInverseCellWidth,
    gridCellWidth,
    dev_gridCellStartIndices,
    dev_gridCellEndIndices,
    dev_sorted_pos,
    dev_vel1,
    dev_vel2
    );

    //  Update position
  kernUpdatePos << <fullBlocksPerGrid, blockSize >> >(
    numObjects,
    dt,
    dev_sorted_pos,
    dev_vel2
    );

  // Ping pong
  std::swap(dev_pos, dev_sorted_pos);
}

void Boids::endSimulation() {
  hipFree(dev_vel1);
  hipFree(dev_vel2);
  hipFree(dev_pos);

  // TODO-2.1 TODO-2.3 - Free any additional buffers here.
  hipFree(dev_particleArrayIndices);
  hipFree(dev_particleGridIndices);
  hipFree(dev_gridCellStartIndices);
  hipFree(dev_gridCellEndIndices);
  hipFree(dev_sorted_pos);
  hipFree(dev_sorted_vel1);
  hipFree(dev_sorted_vel2);
}

void Boids::unitTest() {
  // LOOK-1.2 Feel free to write additional tests here.

  // test unstable sort
  int *dev_intKeys;
  int *dev_intValues;
  int N = 10;

  int *intKeys = new int[N];
  int *intValues = new int[N];

  intKeys[0] = 0; intValues[0] = 0;
  intKeys[1] = 1; intValues[1] = 1;
  intKeys[2] = 0; intValues[2] = 2;
  intKeys[3] = 3; intValues[3] = 3;
  intKeys[4] = 0; intValues[4] = 4;
  intKeys[5] = 2; intValues[5] = 5;
  intKeys[6] = 2; intValues[6] = 6;
  intKeys[7] = 0; intValues[7] = 7;
  intKeys[8] = 5; intValues[8] = 8;
  intKeys[9] = 6; intValues[9] = 9;

  hipMalloc((void**)&dev_intKeys, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intKeys failed!");

  hipMalloc((void**)&dev_intValues, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intValues failed!");

  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  std::cout << "before unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // How to copy data to the GPU
  hipMemcpy(dev_intKeys, intKeys, sizeof(int) * N, hipMemcpyHostToDevice);
  hipMemcpy(dev_intValues, intValues, sizeof(int) * N, hipMemcpyHostToDevice);

  // ------------

  // Wrap device vectors in thrust iterators for use with thrust.
  thrust::device_ptr<int> dev_thrust_keys(dev_intKeys);
  thrust::device_ptr<int> dev_thrust_values(dev_intValues);
  // LOOK-2.1 Example for using thrust::sort_by_key
  thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + N, dev_thrust_values);

  // How to copy data back to the CPU side from the GPU
  hipMemcpy(intKeys, dev_intKeys, sizeof(int) * N, hipMemcpyDeviceToHost);
  hipMemcpy(intValues, dev_intValues, sizeof(int) * N, hipMemcpyDeviceToHost);
  checkCUDAErrorWithLine("memcpy back failed!");

  std::cout << "after unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // ------------
  float test_cellWidth = 10.0;
  float test_sceneScale = 100.0f;
  int test_sideCount = (int)(test_sceneScale / test_cellWidth);
  int test_halfSideCount = test_sideCount / 2;
  int test_gridCellCount = test_sideCount * test_sideCount * test_sideCount;
  float test_inverseCellWidth = 1.0f / test_cellWidth;
  float test_halfGridWidth = test_cellWidth * test_halfSideCount;
  glm::vec3 test_gridMinimum(-test_halfGridWidth, -test_halfGridWidth, -test_halfGridWidth);

  std::cout << "Test cell width: " << test_cellWidth << std::endl;
  std::cout << "Test scene scale: " << test_sceneScale << std::endl;
  std::cout << "Test half side count: " << test_halfSideCount << std::endl;
  std::cout << "Test side count: " << test_sideCount << std::endl;
  std::cout << "Test grid cell count: " << test_gridCellCount << std::endl;
  std::cout << "Test inverse cell width: " << test_inverseCellWidth << std::endl;
  std::cout << "Test half grid width: " << test_halfGridWidth << std::endl;
  std::cout << "Test grid minimum: " << test_gridMinimum.x << ", " << test_gridMinimum.y << ", " << test_gridMinimum.z << std::endl;

  int* test_particleArrayIndices = new int[N];
  int* test_particleGridIndices = new int[N];
  glm::vec3* test_pos = new glm::vec3[N];

  glm::vec3* test_dev_pos;
  int* test_dev_particleArrayIndices;
  int* test_dev_particleGridIndices;

  hipMalloc((void**)&test_dev_pos, N * sizeof(glm::vec3));
  for (int i = 0; i < N; ++i) {
    test_pos[i] = glm::vec3(test_cellWidth * i) + test_gridMinimum;

  }
  hipMemcpy(test_dev_pos, test_pos, sizeof(glm::vec3) * N, hipMemcpyHostToDevice);

  hipMalloc((void**)&test_dev_particleArrayIndices, N * sizeof(int));
  hipMalloc((void**)&test_dev_particleGridIndices, N * sizeof(int));

  kernComputeIndices << <1, N >> >(
    N, 
      test_sideCount,
        test_gridMinimum,
          test_inverseCellWidth,
            test_dev_pos,
              test_dev_particleArrayIndices,
                test_dev_particleGridIndices
                  );

  hipMemcpy(test_pos, test_dev_pos, sizeof(glm::vec3) * N, hipMemcpyDeviceToHost);
  hipMemcpy(test_particleArrayIndices, test_dev_particleArrayIndices, sizeof(int) * N, hipMemcpyDeviceToHost);
  hipMemcpy(test_particleGridIndices, test_dev_particleGridIndices, sizeof(int) * N, hipMemcpyDeviceToHost);

  std::cout << "Test kernComputeIndices  : " << std::endl;
  for (auto i = 0; i < N; i++) {
    std::cout << "  pos: " << test_pos[i].x << ", " << test_pos[i].y << ", " << test_pos[i].z;
    std::cout << "  particle array indices: " << test_particleArrayIndices[i];
    std::cout << " particle grid indices: " << test_particleGridIndices[i] << std::endl;
  }

  // ------------

  int *dev_keyStarts;
  int *dev_keyEnds;
  hipMalloc((void**)&dev_keyStarts, N * sizeof(int));
  hipMalloc((void**)&dev_keyEnds, N * sizeof(int));

  int *intStarts = new int[N];
  int *intEnds = new int[N];

  kernResetIntBuffer << <1, N >> >(N, dev_keyStarts, INVALID_VALUE);
  kernResetIntBuffer << <1, N >> >(N, dev_keyEnds, INVALID_VALUE);
  hipMemcpy(intStarts, dev_keyStarts, sizeof(int) * N, hipMemcpyDeviceToHost);
  hipMemcpy(intEnds, dev_keyEnds, sizeof(int) * N, hipMemcpyDeviceToHost);

  std::cout << "Test kernResetIntBuffer  : " << std::endl;
  for (auto i = 0; i < N; i++) {
    std::cout << "  start: " << intStarts[i];
    std::cout << " end: " << intEnds[i] << std::endl;
  }

  // ------------

  kernIdentifyCellStartEnd << <1, N >> >(N, dev_intKeys, dev_keyStarts, dev_keyEnds);

  hipMemcpy(intStarts, dev_keyStarts, sizeof(int) * N, hipMemcpyDeviceToHost);
  hipMemcpy(intEnds, dev_keyEnds, sizeof(int) * N, hipMemcpyDeviceToHost);

  std::cout << "Test kernIdentifyCellStartEnd  : " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  start: " << intStarts[i];
    std::cout << " end: " << intEnds[i] << std::endl;
  }

  // --------------
  glm::vec3* test_sorted_pos = new glm::vec3[N];

  glm::vec3* test_dev_sorted_pos;
  hipMalloc((void**)&test_dev_sorted_pos, N * sizeof(glm::vec3));

  std::cout << "Test kernRearrangeBuffer  : " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  pos: " << test_pos[i].x << ", " << test_pos[i].y << ", " << test_pos[i].z;
    std::cout << " index: " << intValues[i] << std::endl;
  }
  kernRearrangeBuffer << <1, N >> >(N, dev_intValues, test_dev_sorted_pos, test_dev_pos);
  hipMemcpy(test_sorted_pos, test_dev_sorted_pos, sizeof(glm::vec3) * N, hipMemcpyDeviceToHost);

  for (int i = 0; i < N; i++) {
    std::cout << "  sorted_pos: " << test_sorted_pos[i].x << ", " << test_sorted_pos[i].y << ", " << test_sorted_pos[i].z;
    std::cout << " index: " << intValues[i] << std::endl;
  }


  // cleanup
  delete[] intKeys;
  delete[] intValues;
  delete[] intStarts;
  delete[] intEnds;
  delete[] test_particleArrayIndices;
  delete[] test_particleGridIndices;
  delete[] test_pos;
  delete[] test_sorted_pos;
  hipFree(dev_intKeys);
  hipFree(dev_intValues);
  hipFree(dev_keyStarts);
  hipFree(dev_keyEnds);
  hipFree(test_dev_pos);
  hipFree(test_dev_particleArrayIndices);
  hipFree(test_dev_particleGridIndices);
  hipFree(test_dev_sorted_pos);
  checkCUDAErrorWithLine("hipFree failed!");
  return;
}
